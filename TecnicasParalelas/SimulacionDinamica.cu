#include <iostream>
#include <hip/hip_runtime.h>

#define N 2048  // Tamaño de la cuadrícula
#define DT 0.1f  // Paso de tiempo
#define VISC 0.0001f  // Viscosidad del fluido
#define ITERACIONES 1000  // Más iteraciones para notar diferencia

// Kernel para difundir la velocidad del fluido
__global__ void diffuse(float *vx, float *vy, float *vx_new, float *vy_new) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * N + x;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1) {
        vx_new[index] = vx[index] + VISC * DT * (
            vx[index - 1] + vx[index + 1] + vx[index - N] + vx[index + N] - 4 * vx[index]);
        vy_new[index] = vy[index] + VISC * DT * (
            vy[index - 1] + vy[index + 1] + vy[index - N] + vy[index + N] - 4 * vy[index]);
    }
}

// Kernel para advectar la velocidad del fluido
__global__ void advect(float *vx, float *vy, float *vx_new, float *vy_new) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * N + x;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1) {
        float x_prev = x - DT * vx[index];
        float y_prev = y - DT * vy[index];

        int x0 = (int)x_prev, y0 = (int)y_prev;
        int x1 = x0 + 1, y1 = y0 + 1;

        vx_new[index] = 0.5f * (vx[y0 * N + x0] + vx[y1 * N + x1]);
        vy_new[index] = 0.5f * (vy[y0 * N + x0] + vy[y1 * N + x1]);
    }
}

int main() {
    float *vx, *vy, *vx_new, *vy_new;
    float *d_vx, *d_vy, *d_vx_new, *d_vy_new;
    
    size_t size = N * N * sizeof(float);
    vx = new float[N * N]();
    vy = new float[N * N]();
    vx_new = new float[N * N]();
    vy_new = new float[N * N]();

    hipMalloc(&d_vx, size);
    hipMalloc(&d_vy, size);
    hipMalloc(&d_vx_new, size);
    hipMalloc(&d_vy_new, size);

    hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < ITERACIONES; i++) {
        diffuse<<<numBlocks, threadsPerBlock>>>(d_vx, d_vy, d_vx_new, d_vy_new);
        advect<<<numBlocks, threadsPerBlock>>>(d_vx_new, d_vy_new, d_vx, d_vy);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "🔥 Tiempo de ejecución en GPU: " << milliseconds << " ms" << std::endl;

    hipMemcpy(vx, d_vx, size, hipMemcpyDeviceToHost);
    hipMemcpy(vy, d_vy, size, hipMemcpyDeviceToHost);

    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_vx_new);
    hipFree(d_vy_new);
    delete[] vx;
    delete[] vy;
    delete[] vx_new;
    delete[] vy_new;
    return 0;
}
